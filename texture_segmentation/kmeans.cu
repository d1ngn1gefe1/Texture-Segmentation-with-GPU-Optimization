#include "hip/hip_runtime.h"
#include "kmeans.h"

// Helpoer functions for Kmean
static inline int nextPowerOfTwo(int n) {
    n--;
    n = n >>  1 | n;
    n = n >>  2 | n;
    n = n >>  4 | n;
    n = n >>  8 | n;
    n = n >> 16 | n;
//  n = n >> 32 | n;    //  For 64-bit ints
    return ++n;
}

__host__ __device__ inline static
float euclid_dist_2(int    numCoords,
                    int    numObjs,
                    int    numClusters,
                    float *objects,     // [numDims][numObjs]
                    float *clusters,    // [numDims][numClusters]
                    int    objectId,
                    int    clusterId)
{
    float ans=0.0f;
    for (int i = 0; i < numCoords; i++) {
        ans += (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]) *
               (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]);
    }
    return(ans);
}


__global__ static
void findNearestCluster(int numDims, int numObjs, int numClusters,
                          float *objects,           //  [numDims][numObjs]
                          float *deviceClusters,    //  [numDims][numClusters]
                          int *membership,          //  [numObjs]
                          int *intermediates)
{
    extern __shared__ char sharedMemory[];

    //  The type chosen for membershipChanged must be large enough to support
    //  reductions! There are blockDim.x elements, one for each thread in the
    //  block. See numThreadsPerClusterBlock in cuda_kmeans().
    unsigned char *membershipChanged = (unsigned char *)sharedMemory;

    float *clusters = (float *)(sharedMemory + blockDim.x);


    membershipChanged[threadIdx.x] = 0;

    //  BEWARE: We can overrun our shared memory here if there are too many
    //  clusters or too many coordinates! For reference, a Tesla C1060 has 16
    //  KiB of shared memory per block, and a GeForce GTX 480 has 48 KiB of
    //  shared memory per block.
    for (int i = threadIdx.x; i < numClusters; i += blockDim.x) {
        for (int j = 0; j < numDims; j++) {
            clusters[numClusters * j + i] = deviceClusters[numClusters * j + i];
        }
    }
    __syncthreads();


    int objectId = blockDim.x * blockIdx.x + threadIdx.x;

    if (objectId < numObjs) {
        int   index, i;
        float dist, min_dist;

        /* find the cluster id that has min distance to object */
        index    = 0;
        min_dist = euclid_dist_2(numDims, numObjs, numClusters,
                                 objects, clusters, objectId, 0);

        for (i=1; i<numClusters; i++) {
            dist = euclid_dist_2(numDims, numObjs, numClusters,
                                 objects, clusters, objectId, i);
            /* no need square root */
            if (dist < min_dist) { /* find the min and its array index */
                min_dist = dist;
                index    = i;
            }
        }

        if (membership[objectId] != index) {
            membershipChanged[threadIdx.x] = 1;
        }

        /* assign the membership to object objectId */
        membership[objectId] = index;

        __syncthreads();    //  For membershipChanged[]

        //  blockDim.x *must* be a power of two!
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                membershipChanged[threadIdx.x] +=
                    membershipChanged[threadIdx.x + s];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            intermediates[blockIdx.x] = membershipChanged[0];
        }
    }
}


__global__ static
void computeDelta(int *deviceIntermediates,
                   int numIntermediates,    //  The actual number of intermediates
                   int numIntermediates2)   //  The next power of two
{
    //  The number of elements in this array should be equal to
    //  numIntermediates2, the number of threads launched. It *must* be a power
    //  of two!
    extern __shared__ unsigned int intermediates[];

    //  Copy global intermediate values into shared memory.
    intermediates[threadIdx.x] =
        (threadIdx.x < numIntermediates) ? deviceIntermediates[threadIdx.x] : 0;

    __syncthreads();

    //  numIntermediates2 *must* be a power of two!
    for (unsigned int s = numIntermediates2 / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            intermediates[threadIdx.x] += intermediates[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        deviceIntermediates[0] = intermediates[0];
    }
}

/* return an array of cluster centers of size [numClusters][numCoords]       */
void cuda_kmeans(float **objects,      /* in: [numObjs][numCoords] */
                   int     numDims,    /* no. features */
                   int     numObjs,      /* no. objects */
                   int     numClusters,  /* no. clusters */
                   float   threshold,    /* % objects change membership */
                   int    *membership,   /* out: [numObjs] */
                   int    *loop_iterations,
				   float  **clusters
				   )
{
    int i, j, index, loop=0;
    int *newClusterSize; /* [numClusters]: no. objects assigned in each
                                new cluster */
    float delta;          /* % of objects change their clusters */
    float  **dimObjects;
    //float  **clusters;       /* out: [numClusters][numCoords] */
    float  **dimClusters;
    float  **newClusters;    /* [numCoords][numClusters] */

    float *deviceObjects;
    float *deviceClusters;
    int *deviceMembership;
    int *deviceIntermediates;

    //  Copy objects given in [numObjs][numCoords] layout to new
    //  [numDims][numObjs] layout
    malloc2D(dimObjects, numDims, numObjs, float);
    for (i = 0; i < numDims; i++) {
        for (j = 0; j < numObjs; j++) {
            dimObjects[i][j] = objects[j][i];
        }
    }

    /* pick first numClusters elements of objects[] as initial cluster centers*/
    malloc2D(dimClusters, numDims, numClusters, float);
    for (i = 0; i < numDims; i++) {
        for (j = 0; j < numClusters; j++) {
            dimClusters[i][j] = clusters[j][i];
        }
    }

    /* initialize membership[] */
    for (i=0; i<numObjs; i++) membership[i] = -1;

    /* need to initialize newClusterSize and newClusters[0] to all 0 */
    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL);

    malloc2D(newClusters, numDims, numClusters, float);
    memset(newClusters[0], 0, numDims * numClusters * sizeof(float));

    //  To support reduction, numThreadsPerClusterBlock *must* be a power of
    //  two, and it *must* be no larger than the number of bits that will
    //  fit into an unsigned char, the type used to keep track of membership
    //  changes in the kernel.
    const unsigned int numThreadsPerClusterBlock = 128;
    const unsigned int numClusterBlocks =
        (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock;

    const unsigned int clusterBlockSharedDataSize =
        numThreadsPerClusterBlock * sizeof(unsigned char) +
        numClusters * numDims * sizeof(float);

    hipDeviceProp_t deviceProp;
    int deviceNum;
    hipGetDevice(&deviceNum);
    hipGetDeviceProperties(&deviceProp, deviceNum);

    if (clusterBlockSharedDataSize > deviceProp.sharedMemPerBlock) {
        err("WARNING: Your CUDA hardware has insufficient block shared memory. "
            "You need to recompile with BLOCK_SHARED_MEM_OPTIMIZATION=0. "
            "See the README for details.\n");
    }

    const unsigned int numReductionThreads =
        nextPowerOfTwo(numClusterBlocks);
    const unsigned int reductionBlockSharedDataSize =
        numReductionThreads * sizeof(unsigned int);

    checkCuda(hipMalloc(&deviceObjects, numObjs*numDims*sizeof(float)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numDims*sizeof(float)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&deviceIntermediates, numReductionThreads*sizeof(unsigned int)));

    checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
              numObjs*numDims*sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
              numObjs*sizeof(int), hipMemcpyHostToDevice));

    do {
        checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
                  numClusters*numDims*sizeof(float), hipMemcpyHostToDevice));

        findNearestCluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numDims, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, deviceIntermediates);

        hipDeviceSynchronize(); checkLastCudaError();

        computeDelta <<< 1, numReductionThreads, reductionBlockSharedDataSize >>>
            (deviceIntermediates, numClusterBlocks, numReductionThreads);

        hipDeviceSynchronize(); checkLastCudaError();

        int d;
        checkCuda(hipMemcpy(&d, deviceIntermediates,
                  sizeof(int), hipMemcpyDeviceToHost));
        delta = (float)d;

        checkCuda(hipMemcpy(membership, deviceMembership,
                  numObjs*sizeof(int), hipMemcpyDeviceToHost));

        for (i=0; i<numObjs; i++) {
            /* find the array index of nestest cluster center */
            index = membership[i];

            /* update new cluster centers : sum of objects located within */
            newClusterSize[index]++;
            for (j=0; j<numDims; j++)
                newClusters[j][index] += objects[i][j];
        }

        //  TODO: Flip the nesting order
        //  TODO: Change layout of newClusters to [numClusters][numCoords]
        /* average the sum and replace old cluster centers with newClusters */
        for (i=0; i<numClusters; i++) {
            for (j=0; j<numDims; j++) {
                if (newClusterSize[i] > 0)
                    dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
                newClusters[j][i] = 0.0;   /* set back to 0 */
            }
            newClusterSize[i] = 0;   /* set back to 0 */
        }

        delta /= numObjs;
    } while (delta > threshold && loop++ < 100);

    *loop_iterations = loop + 1;

    /* allocate a 2D space for returning variable clusters[] (coordinates
       of cluster centers) */
    //malloc2D(clusters, numClusters,numDims, float);
    for (i = 0; i < numClusters; i++) {
        for (j = 0; j < numDims; j++) {
            clusters[i][j] = dimClusters[j][i];
        }
    }

    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(deviceMembership));
    checkCuda(hipFree(deviceIntermediates));

    free(dimObjects[0]);
    free(dimObjects);
    free(dimClusters[0]);
    free(dimClusters);
    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    //return clusters;
}
